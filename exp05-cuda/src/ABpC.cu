#include <cstdint>
#include <hip/hip_runtime.h>

extern "C"
{
    void cudaABpC(float* A, float* B, float* C, size_t a, size_t b, size_t c);
}

/**
 * @brief Matrix multiplication: C = A * B.
 *
 * @param A Matrix of shape (a, b).
 * @param B Matrix of shape (b, c).
 * @param C Matrix of shape (a, c).
 */
__global__ void cudaABpC_kernel(float* A, float* B, float* C, size_t a, size_t b, size_t c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < a && j < c) {
        float sum = 0.0f;
        for (int k = 0; k < b; k++) {
            sum += A[i * b + k] * B[k * c + j];
        }
        C[i * c + j] = sum;
    }
}

void cudaABpC(float* A, float* B, float* C, size_t a, size_t b, size_t c)
{
    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, a * b * sizeof(float));
    hipMalloc(&d_B, b * c * sizeof(float));
    hipMalloc(&d_C, a * c * sizeof(float));

    hipMemcpy(d_A, A, a * b * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, b * c * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((a + block.x - 1) / block.x, (c + block.y - 1) / block.y);

    cudaABpC_kernel<<<grid, block>>>(d_A, d_B, d_C, a, b, c);

    hipMemcpy(C, d_C, a * c * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}