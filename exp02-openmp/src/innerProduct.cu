#include <hip/hip_runtime.h>

#include <cstdio>

extern "C"
{
    void cuInnerProduct(float* a, float* b, float* c, int n);
}

__global__ void cuInnerProductKernel(float* a, float* b, float* c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}

void cuInnerProduct(float* a, float* b, float* c, int n)
{
    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 512;
    int numBlocks = (n + blockSize - 1) / blockSize;

    cuInnerProductKernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}