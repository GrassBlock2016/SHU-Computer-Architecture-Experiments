#include "hip/hip_runtime.h"
#include "InnerProduct.hpp"
#include <cstdio>

__global__ void cuInnerProductKernel(float* a, float* b, float* c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}

void cuInnerProduct(float* a, float* b, float* c, int n)
{
    float *d_a, *d_b, *d_c;

    hipError_t err;
    err = hipMalloc(&d_a, n * sizeof(float));
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMalloc(&d_b, n * sizeof(float));
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMalloc(&d_c, n * sizeof(float));
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }

    err = hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }

    int blockSize = 512;
    int numBlocks = (n + blockSize - 1) / blockSize;

    cuInnerProductKernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    err = hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }

    err = hipFree(d_a);
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(d_b);
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }
    err = hipFree(d_c);
    if (err != hipSuccess) {
        printf("[CUDA ERROR]: %s\n", hipGetErrorString(err));
        return;
    }
}